#include "hip/hip_runtime.h"
#include "common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define BDIMX 16

void printData(int *in, int size) {
	for (int i = 0; i < size; i++) printf("%2d ", in[i]);
	printf("\n");
}

__global__ void test_shfl_wrap(int *dout, int *din, int offset) {
	int value = din[threadIdx.x];
	value = __shfl(value, threadIdx.x + offset, BDIMX);
	dout[threadIdx.x] = value;
}

__global__ void test_shfl_wrap_plus(int *dout, int *din, int offset) {
	int value = din[threadIdx.x];
	value += __shfl(value, threadIdx.x + offset, BDIMX);
	dout[threadIdx.x] = value;
}

int main(int argc, char **argv) {
	int dev = 0;
	bool iPrintout = 1;

	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("> %s Starting.", argv[0]);
	printf("at Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	int nElem = BDIMX;
	int h_din[BDIMX], h_dout[BDIMX];

	for (int i = 0; i < nElem; i++) {
		h_din[i] = i;
	}

	if (iPrintout) {
		printf("Initial data\t\t: ");
		printData(h_din, nElem);
	}

	size_t nBytes = nElem * sizeof(int);
	int *d_din, *d_dout;
	CHECK(hipMalloc((int**)&d_din, nBytes));
	CHECK(hipMalloc((int**)&d_dout, nBytes));

	CHECK(hipMemcpy(d_din, h_din, nBytes, hipMemcpyHostToDevice));

	int block = BDIMX;

	test_shfl_wrap<<<1, block>>>(d_dout, d_din, -2);
	CHECK(hipGetLastError());
	CHECK(hipMemcpy(h_dout, d_dout, nBytes, hipMemcpyDeviceToHost));
	if (iPrintout) {
		printf("test_shfl_wrap\t\t: ");
		printData(h_dout, nElem);
	}

	test_shfl_wrap_plus<<<1, block>>>(d_dout, d_din, 2);
	CHECK(hipGetLastError());
	CHECK(hipMemcpy(h_dout, d_dout, nBytes, hipMemcpyDeviceToHost));
	if (iPrintout) {
		printf("test_shfl_wrap_plus\t: ");
		printData(h_dout, nElem);
	}

	CHECK(hipFree(d_din));
	CHECK(hipFree(d_dout));
	CHECK(hipDeviceReset());

	return EXIT_SUCCESS;
}
