#include<hip/hip_runtime.h>
#include<stdio.h>

#include <hip/hip_runtime_api.h>

#define NSTREAM 8

void matricMul(int *A, int *B, int *C, int size) {
	for (int col = 0; col < size; col++) {
		for (int row = 0; row < size; row++) {
			int outidx = col * size + row;
			for (int idx = 0; idx < size; idx++) {
				C[outidx] += A[col*size + idx] * B[idx*size + row];
			}
		}
	}
}

void matrixMulCheck(int *C_cpu, int *C_gpu, int size) {
	bool ResultFlag = true;

	// Print the result
	for (int i = 0; i < size; i++) {
		if (C_cpu[i] != C_gpu[i]) {
			ResultFlag = false;
			printf("Error: C_cpu[%d] = %d; C_gpu[%d] = %d;\n", i, C_cpu[i], i, C_gpu[i]);
			break;
		}
	}
	if (ResultFlag == true) printf("Matrix Multiplication OK!\n");
	else printf("Matrix Multiplication Error!\n");
}

__global__ void matrixMulDepth(int *A, int *B, int *C, int size) {
	int tid, tx, ty;
	tx = threadIdx.x + blockDim.x * blockIdx.x;
	ty = threadIdx.y + blockDim.y * blockIdx.y;
	tid = size * ty + tx;

	int Aval = 0;
	int Bval = 0;
	int Cval = 0;

	for (int i = 0; i < size; i++) {
		Aval = A[ty * size + i];
		Bval = B[i * size + tx];
		Cval += Aval * Bval;
	}

	C[tid] = Cval;
}

int main() {
	int nx = 1600;
	int ny = 1600;
	int dimx = 32;
	int dimy = 16;

	dim3 block(dimx, dimy);	// Block dimension 32x16
	dim3 grid((nx+block.x-1)/block.x, (ny/NSTREAM+block.y-1)/block.y);

	char *iname = "CUDA_DEVICE_MAX_CONNECTIONS";
	setenv(iname, "4", 1);

	int MatrixSize = nx * ny;
	int BufferSize = MatrixSize * sizeof(int);
	int iElem = MatrixSize / NSTREAM;
	int ibytes = iElem * sizeof(int);

	int *h_A, *h_B, *h_C;
	int *C_cpu;

	// Create streams
	hipStream_t *stream = (hipStream_t*)malloc(NSTREAM*sizeof(hipStream_t));
	for (int i = 0; i < NSTREAM; i++) {
		hipStreamCreate(&stream[i]);
	}

	// Host memory allocation
	hipHostAlloc((void**)&h_A, BufferSize, hipHostMallocDefault);
	hipHostAlloc((void**)&h_B, BufferSize, hipHostMallocDefault);
	hipHostAlloc((void**)&h_C, BufferSize, hipHostMallocDefault);
	hipHostAlloc((void**)&C_cpu, BufferSize, hipHostMallocDefault);

	// Data input
	for (int i = 0; i < MatrixSize; i++) {
		h_A[i] = i % 100;
		h_B[i] = i % 100;
		h_C[i] = 0;
		C_cpu[i] = 0;
	}

	int *d_A, *d_B, *d_C;

	// Device memory allocation
	hipMalloc((void**)&d_A, BufferSize);
	hipMalloc((void**)&d_B, BufferSize);
	hipMalloc((void**)&d_C, BufferSize);

	hipHostGetDevicePointer((void**)&d_B, (void*)h_B, 0);

	for (int i = 0; i < NSTREAM; i++) {
		int ioffset = i * iElem;
		hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], ibytes, hipMemcpyHostToDevice, stream[i]);
		matrixMulDepth<<<grid, block, 0, stream[i]>>>(&d_A[ioffset], d_B, &d_C[ioffset], nx);
		hipMemcpyAsync(&h_C[ioffset], &d_C[ioffset], ibytes, hipMemcpyDeviceToHost, stream[i]);
		hipStreamSynchronize(stream[i]);
	}

	hipProfilerStop();

	// Check result
	matricMul(h_A, h_B, C_cpu, nx);
	matrixMulCheck(C_cpu, h_C, nx);

	// Free device memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// Free host memory
	free(h_A);
	free(h_B);
	free(h_C);
	free(C_cpu);
	
	return 0;
}