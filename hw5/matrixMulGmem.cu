#include<hip/hip_runtime.h>
#include<stdio.h>


void matricMul(int *A, int *B, int *C, int size) {
	for (int col = 0; col < size; col++) {
		for (int row = 0; row < size; row++) {
			int outidx = col * size + row;
			for (int idx = 0; idx < size; idx++) {
				C[outidx] += A[col*size + idx] * B[idx*size + row];
			}
		}
	}
}

void matrixMulCheck(int *C_cpu, int *C_gpu, int size) {
	bool ResultFlag = true;

	// Print the result
	for (int i = 0; i < size; i++) {
		if (C_cpu[i] != C_gpu[i]) {
			ResultFlag = false;
			printf("Error: C_cpu[%d] = %d; C_gpu[%d] = %d;\n", i, C_cpu[i], i, C_gpu[i]);
			break;
		}
	}
	if (ResultFlag == true) printf("Matrix Multiplication OK!\n");
	else printf("Matrix Multiplication Error!\n");
}

__global__ void matrixMulGmem(int *A, int *B, int *C, int size) {
	int tid, tx, ty;
	tx = threadIdx.x + blockDim.x * blockIdx.x;
	ty = threadIdx.y + blockDim.y * blockIdx.y;
	tid = size * ty + tx;

	int Aval = 0;
	int Bval = 0;
	int Cval = 0;

	for (int i = 0; i < size; i++) {
		Aval = A[ty * size + i];
		Bval = B[i * size + tx];
		Cval += Aval * Bval;
	}

	C[tid] = Cval;
}

int main() {
	int nx = 1600;
	int ny = 1600;
	int dimx = 32;
	int dimy = 16;

	dim3 block(dimx, dimy);	// Block dimension 32x16
	dim3 grid((nx+block.x-1)/block.x, (ny+block.y-1)/block.y);

	int MatrixSize = nx * ny;
	int BufferSize = MatrixSize * sizeof(int);

	// Create events and streams
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int *h_A, *h_B, *h_C;
	int *C_cpu;

	// Host memory allocation
	h_A = (int*)malloc(BufferSize);
	h_B = (int*)malloc(BufferSize);
	h_C = (int*)malloc(BufferSize);
	C_cpu = (int*)malloc(BufferSize);

	// Data input
	for (int i = 0; i < MatrixSize; i++) {
		h_A[i] = i % 100;
		h_B[i] = i % 100;
		h_C[i] = 0;
		C_cpu[i] = 0;
	}

	int *d_A, *d_B, *d_C;

	// Device memory allocation
	hipMalloc((void**)&d_A, BufferSize);
	hipMalloc((void**)&d_B, BufferSize);
	hipMalloc((void**)&d_C, BufferSize);

	hipEventRecord(start);

	// Copy data from Host to Device
	hipMemcpy(d_A, h_A, BufferSize, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, BufferSize, hipMemcpyHostToDevice);

	// Matrix Multiplication using only the global memory
	matrixMulGmem<<<grid, block>>>(d_A, d_B, d_C, nx);

	// Copy result from Device to Host
	hipMemcpy(h_C, d_C, BufferSize, hipMemcpyDeviceToHost);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	
	float time;
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Check result
	matricMul(h_A, h_B, C_cpu, nx);
	matrixMulCheck(C_cpu, h_C, nx);

	// Free memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(h_A);
	free(h_B);
	free(h_C);
	free(C_cpu);

	return 0;
}