#include <stdio.h>
#include <hip/hip_runtime.h>

void MatrixAddC(float* A, float* B, float* S, int Width, int Height, int offset) {
	int col = 0;
	int row = 0;
	int DestIndex = 0;
	int N = Width * Height;
	for (col = 0; col < Width; col++) {
		for (row = 0; row < Height; row++) {
			DestIndex = col * Width + row;
			S[DestIndex] = A[DestIndex + offset] + B[DestIndex + offset];
		}
	}
}

__global__ void MatrixAddGlobalMem(float* A, float* B, float* S, int Width, int Height, int offset) {
	int tid, tx, ty, N;
	tx = threadIdx.x + blockIdx.x * blockDim.x;
	ty = threadIdx.y + blockIdx.y * blockDim.y;
	tid = Width * ty + tx;
	N = Width * Height;
	int k = tid + offset;
	if (k < N) S[tid] = A[k] + B[k];
}

int main() {
	int nx = 1600;
	int ny = 1600;
	int dimx = 32;
	int dimy = 32;

	int offset = 0;

	dim3 block(dimx, dimy);
	dim3 grid((nx+block.x-1)/block.x, (ny+block.y-1)/block.y);

	const int MatrixSize = nx * ny;
	const int BufferSize = MatrixSize * sizeof(float);

	float* A;
	float* B;
	float* Sum;
	float* S_C;

	// Host memory allocation
	A = (float*)malloc(BufferSize);
	B = (float*)malloc(BufferSize);
	Sum = (float*)malloc(BufferSize);
	S_C = (float*)malloc(BufferSize);

	// Data input
	for (int i = 0; i < MatrixSize; i++) {
		A[i] = i;
		B[i] = i;
		Sum[i] = 0;
	}

	float* dev_A;
	float* dev_B;
	float* dev_S;

	// Device memory allocation
	hipMalloc((float**)&dev_A, BufferSize);
	hipMalloc((float**)&dev_B, BufferSize);
	hipMalloc((float**)&dev_S, BufferSize);

	// Copy host to device
	hipMemcpy(dev_A, A, BufferSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B, BufferSize, hipMemcpyHostToDevice);

	// Kernel function
	MatrixAddGlobalMem <<<grid, block>>> (dev_A, dev_B, dev_S, nx, ny, offset);

	// Copy result from Device to Host
	hipMemcpy(Sum, dev_S, BufferSize, hipMemcpyDeviceToHost);

	MatrixAddC(A, B, S_C, nx, ny, offset);
	bool ResultFlag = true;
	for (int i = 0; i < MatrixSize; i++) {
		if (Sum[i] != S_C[i]) {
			ResultFlag = false;
			printf("Error Matrix Add at element %d\n", i);
			break;
		}
	}
	if (ResultFlag == true) printf("Matrix Add is OK\n");
	else printf("Error Matrix Add\n");

	// Free memory
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_S);
	
	free(A);
	free(B);
	free(Sum);
	free(S_C);
}
