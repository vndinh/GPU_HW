#include <stdio.h>
#include <hip/hip_runtime.h>


#define TILE_WIDTH 32

void matricMul(int *A, int *B, int *C, int size) {
	for (int col = 0; col < size; col++) {
		for (int row = 0; row < size; row++){
			int outidx = col * size + row;
			for (int idx = 0; idx < size; idx++)
				C[outidx] += A[col*size+idx] * B[idx*size+row];
		}
	}
}

void matrixMulCheck(int *C_test, int *C_cuda, int size) {
	bool ResultFlag = true;

	// Print the result
	for (int i = 0; i < size; i++) {
		if (C_test[i] != C_cuda[i]) {
			ResultFlag = false;
			printf("Error: C_test[%d] = %d; C_cuda[%d] = %d;\n", i, C_test[i], i, C_cuda[i]);
			break;
		}
	}
	if (ResultFlag == true) printf("Matrix Multiplication OK!\n");
	else printf("Matrix Multiplication Error!\n");
}

__global__ void matrixMulGSmem(int *A, int *B, int *C, int size) {

	// Static shared memory
	__shared__ int ds_A[TILE_WIDTH][TILE_WIDTH];
	__shared__ int ds_B[TILE_WIDTH][TILE_WIDTH];

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int row = blockIdx.y * TILE_WIDTH + ty;
	int col = blockIdx.x * TILE_WIDTH + tx;

	int Cval = 0;

	for (int i = 0; i < (size/TILE_WIDTH); i++) {
		if ((row < size) && (i*TILE_WIDTH+tx < size)) ds_A[ty][tx] = A[row*size+i*TILE_WIDTH+tx];
		else ds_A[ty][tx] = 0;

		if ((col < size) && (i*TILE_WIDTH+ty < size)) ds_B[ty][tx] = B[col+size*(i*TILE_WIDTH+ty)];
		else ds_B[ty][tx] = 0;

		__syncthreads();

		for (int j = 0; j < TILE_WIDTH; j++) Cval += ds_A[ty][j] * ds_B[j][tx];

		__syncthreads();
	}

	if (row < size && col < size) C[row * size + col] = Cval;
}

int main() {
	int nx = 1600;
	int ny = 1600;
	int dimx = 32;
	int dimy = 16;

	dim3 block(dimx, dimy);	// Block dimension 32x16
	dim3 grid((nx+block.x-1)/block.x, (ny+block.y-1)/block.y);

	int MatrixSize = nx * ny;
	int BufferSize = MatrixSize * sizeof(int);

	int *h_A;
	int *h_B;
	int *h_C;
	int *C_test;

	// Host memory allocation
	h_A = (int*)malloc(BufferSize);
	h_B = (int*)malloc(BufferSize);
	h_C = (int*)malloc(BufferSize);
	C_test = (int*)malloc(BufferSize);

	// Data input
	for (int i = 0; i < nx; i++) {
		h_A[i] = i % 100;
		h_B[i] = i % 100;
		h_C[i] = 0;
		C_test[i] = 0;
	}

	int *d_A;
	int *d_B;
	int *d_C;

	// Device memory allocation
	hipMalloc((void**)&d_A, BufferSize);
	hipMalloc((void**)&d_B, BufferSize);
	hipMalloc((void**)&d_C, BufferSize);

	// Copy data from Host to Device
	hipMemcpy(d_A, h_A, BufferSize, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, BufferSize, hipMemcpyHostToDevice);

	// Matrix Multiplication
	matrixMulGSmem<<<grid, block, TILE_WIDTH*TILE_WIDTH*sizeof(int)>>>(d_A, d_B, d_C, nx);

	// Copy result from Device to Host
	hipMemcpy(h_C, d_C, BufferSize, hipMemcpyDeviceToHost);

	// Check result
	matricMul(h_A, h_B, C_test, nx);
	matrixMulCheck(C_test, h_C, nx);

	// Free memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(h_A);
	free(h_B);
	free(h_C);
	free(C_test);
	
	return 0;
}
